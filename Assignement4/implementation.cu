/*
============================================================================
Filename    : implementation.cu
Author      : Lucien Michaël Iseli, Loris Pilotto
SCIPER      : 274999, 262651
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>
using namespace std;

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for(int n=0; n<(int) iterations; n++)
    {
        for(int i=1; i<length-1; i++)
        {
            for(int j=1; j<length-1; j++)
            {
                output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                            input[(i-1)*(length)+(j)]   +
                                            input[(i-1)*(length)+(j+1)] +
                                            input[(i)*(length)+(j-1)]   +
                                            input[(i)*(length)+(j)]     +
                                            input[(i)*(length)+(j+1)]   +
                                            input[(i+1)*(length)+(j-1)] +
                                            input[(i+1)*(length)+(j)]   +
                                            input[(i+1)*(length)+(j+1)] ) / 9;

            }
        }
        output[(length/2-1)*length+(length/2-1)] = 1000;
        output[(length/2)*length+(length/2-1)]   = 1000;
        output[(length/2-1)*length+(length/2)]   = 1000;
        output[(length/2)*length+(length/2)]     = 1000;


        temp = input;
        input = output;
        output = temp;
    }
}


__global__
void init_gpu(double* gpu_input, double* gpu_output, int length){
    gpu_input[(length/2-1)*length+(length/2-1)] = 1000;
    gpu_input[(length/2)*length+(length/2-1)]   = 1000;
    gpu_input[(length/2-1)*length+(length/2)]   = 1000;
    gpu_input[(length/2)*length+(length/2)]     = 1000;

    gpu_output[(length/2-1)*length+(length/2-1)] = 1000;
    gpu_output[(length/2)*length+(length/2-1)]   = 1000;
    gpu_output[(length/2-1)*length+(length/2)]   = 1000;
    gpu_output[(length/2)*length+(length/2)]     = 1000;
}

__global__
void compute_gpu(double* gpu_input, double* gpu_output, int length){
    int x_glob = blockIdx.x + 1;
    int y_glob = threadIdx.y + 1;

    if(x_glob == length/2-1 && (y_glob == length/2-1 || y_glob == length/2) ||
       x_glob == length/2 && (y_glob == length/2-1 || y_glob == length/2))
	    return;

    gpu_output[(x_glob)*(length)+(y_glob)] = (gpu_input[(x_glob-1)*(length)+(y_glob-1)] +
                                              gpu_input[(x_glob-1)*(length)+(y_glob)]   +
        	                              gpu_input[(x_glob-1)*(length)+(y_glob+1)] +
                                              gpu_input[(x_glob)*(length)+(y_glob-1)]   +
                                              gpu_input[(x_glob)*(length)+(y_glob)]     +
                                              gpu_input[(x_glob)*(length)+(y_glob+1)]   +
                                              gpu_input[(x_glob+1)*(length)+(y_glob-1)] +
                                              gpu_input[(x_glob+1)*(length)+(y_glob)]   +
                                              gpu_input[(x_glob+1)*(length)+(y_glob+1)] ) /9;
}


// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    //Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);

    /* Preprocessing goes here */
    size_t SIZE = length * length * sizeof(double);
    double* gpu_input;
    double* gpu_output;
    double* temp;
    
    hipMalloc((void**) &gpu_input, SIZE);
    hipMalloc((void**) &gpu_output, SIZE);
    hipMemset(gpu_output, 0, SIZE);
    hipMemset(gpu_input, 0, SIZE);
    /* End preprocessing       */

    hipEventRecord(cpy_H2D_start);
    /* Copying array from host to device goes here */
    init_gpu <<< 1, 1 >>> (gpu_input, gpu_output, length);
    /* End copy array				   */
    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    //Copy array from host to device
    hipEventRecord(comp_start);
    /* GPU calculation goes here */
    dim3 thrsPerBlock(1,length-2);
    dim3 nBlks(length-2,1);

    for(int n = 0; n <(int)iterations; n++){
	compute_gpu <<< nBlks, thrsPerBlock >>> (gpu_input, gpu_output, length);

	temp = gpu_input;
	gpu_input = gpu_output;
	gpu_output = temp;
    }

    /* End GPU calculation	 */
    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);

    hipEventRecord(cpy_D2H_start);
    /* Copying array from device to host goes here */
    hipMemcpy((void*)output,
	       (void*)gpu_output,
	       SIZE,
	       hipMemcpyDeviceToHost);
    /* End copy array 				   */
    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    /* Postprocessing goes here */
    hipFree((void**) &gpu_input);
    hipFree((void**) &gpu_output);

    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout<<"Host to Device MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout<<"Computation takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout<<"Device to Host MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;
}
