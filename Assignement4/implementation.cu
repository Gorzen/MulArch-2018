/*
============================================================================
Filename    : implementation.cu
Author      : Lucien Michaël Iseli, Loris Pilotto
SCIPER      : 274999, 262651
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>
using namespace std;

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for(int n=0; n<(int) iterations; n++)
    {
        for(int i=1; i<length-1; i++)
        {
            for(int j=1; j<length-1; j++)
            {
                output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                            input[(i-1)*(length)+(j)]   +
                                            input[(i-1)*(length)+(j+1)] +
                                            input[(i)*(length)+(j-1)]   +
                                            input[(i)*(length)+(j)]     +
                                            input[(i)*(length)+(j+1)]   +
                                            input[(i+1)*(length)+(j-1)] +
                                            input[(i+1)*(length)+(j)]   +
                                            input[(i+1)*(length)+(j+1)] ) / 9;

            }
        }
        output[(length/2-1)*length+(length/2-1)] = 1000;
        output[(length/2)*length+(length/2-1)]   = 1000;
        output[(length/2-1)*length+(length/2)]   = 1000;
        output[(length/2)*length+(length/2)]     = 1000;

        temp = input;
        input = output;
        output = temp;
    }
}

__global__
void compute_gpu(double* gpu_input, double* gpu_output, int length){
    int x_glob = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y_glob = (blockIdx.y * blockDim.y) + threadIdx.y;

    if(x_glob == length/2-1 && (y_glob == length/2 || y_glob == length/2-1) ||
       x_glob == length/2 && (y_glob == length/2 || y_glob == length/2-1) || 
       x_glob <= 0 || x_glob >= length-1 || y_glob <= 0 || y_glob >= length-1)
	    return;

    gpu_output[(x_glob)*(length)+(y_glob)] = (gpu_input[(x_glob-1)*(length)+(y_glob-1)] +
                                              gpu_input[(x_glob-1)*(length)+(y_glob)]   +
        	                              gpu_input[(x_glob-1)*(length)+(y_glob+1)] +
                                              gpu_input[(x_glob)*(length)+(y_glob-1)]   +
                                              gpu_input[(x_glob)*(length)+(y_glob)]     +
                                              gpu_input[(x_glob)*(length)+(y_glob+1)]   +
                                              gpu_input[(x_glob+1)*(length)+(y_glob-1)] +
                                              gpu_input[(x_glob+1)*(length)+(y_glob)]   +
                                              gpu_input[(x_glob+1)*(length)+(y_glob+1)] ) /9;
}


// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    //Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);

    /* Preprocessing goes here */
    size_t SIZE = length * length * sizeof(double);
    double* gpu_input;
    double* gpu_output;
    double* temp;
    
    hipMalloc((void**) &gpu_input, SIZE);
    hipMalloc((void**) &gpu_output, SIZE);
    /* End preprocessing       */

    hipEventRecord(cpy_H2D_start);
    /* Copying array from host to device goes here */
    hipMemcpy((void*)gpu_input,
	       (void*)input,
	       SIZE,
	       hipMemcpyHostToDevice);
    hipMemcpy((void*)gpu_output,
	       (void*)output,
	       SIZE,
	       hipMemcpyHostToDevice);
    /* End copy array				   */
    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    //Copy array from host to device
    hipEventRecord(comp_start);
    /* GPU calculation goes here */
    dim3 thrsPerBlock(32,32);
    dim3 nBlks(ceil(length/32),ceil(length/32));

    for(int n = 0; n <(int)iterations; n++){
	compute_gpu <<< nBlks, thrsPerBlock >>> (gpu_input, gpu_output, length);
	
	temp = gpu_input;
	gpu_input = gpu_output;
	gpu_output = temp;
    }

    /* End GPU calculation	 */
    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);

    hipEventRecord(cpy_D2H_start);
    /* Copying array from device to host goes here */
    hipMemcpy((void*)output,
	       (void*)gpu_output,
	       SIZE,
	       hipMemcpyDeviceToHost);
    /* End copy array 				   */
    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    /* Postprocessing goes here */
    hipFree((void**) &gpu_input);
    hipFree((void**) &gpu_output);

    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout<<"Host to Device MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout<<"Computation takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout<<"Device to Host MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;
}
